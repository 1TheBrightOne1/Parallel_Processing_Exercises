#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#include <iostream>

__global__ void addVec(int* a, int* b, int* c, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

//3.1 a
__global__
void MatrixAdditionElement(int* left, int* right, int* result, size_t width)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;
    result[index] = left[index] + right[index];
}

//3.1 b 
__global__
void MatrixAdditionRow(int* left, int* right, int* result, size_t width)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < width)
    {
        index *= width;

        for (size_t i = 0; i < width; ++i)
        {
            result[index + i] = left[index + i] + right[index + i];
        }
    }
}

//3.1 c
__global__
void MatrixAdditionCol(int* left, int* right, int* result, size_t width)
{
    size_t elements = width * width;
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < width)
    {
        for (size_t i = index; i < elements; i += blockDim.x)
        {
            result[i] = left[i] + right[i];
        }
    }
}

void Driver3_1(size_t width)
{
    int* A, *B, *C1, *C2, *C3;
    size_t elements = width * width;

    hipMallocManaged(&A, elements * sizeof(int));
    hipMallocManaged(&B, elements * sizeof(int));
    hipMallocManaged(&C1, elements * sizeof(int));
    hipMallocManaged(&C2, elements * sizeof(int));
    hipMallocManaged(&C3, elements * sizeof(int));

    for (size_t i = 0; i < elements; ++i)
    {
        A[i] = i;
        B[i] = elements - i;
    }

    MatrixAdditionElement<<<elements / 256 + 1, 256>>>(A, B, C1, width);
    MatrixAdditionRow<<<width / 256 + 1, 256>>>(A, B, C2, width);
    MatrixAdditionCol<<<width / 256 + 1, 256>>>(A, B, C3, width);

    hipDeviceSynchronize();

    for (size_t i = 0; i < elements; ++i)
    {
        if (C1[i] != C2[i] || C2[i] != C3[i])
        {
            std::cout << "Mismatch at " << i << '\n';
            break;
        }
    }

    std::cout << "Finished\n";

    hipFree(A);
    hipFree(B);
    hipFree(C1);
    hipFree(C2);
    hipFree(C3);
}

int main()
{
    std::cout << "Starting 3_1\nEnter width: ";
    size_t width;
    std::cin >> width;
    Driver3_1(width);

    return 0;
}